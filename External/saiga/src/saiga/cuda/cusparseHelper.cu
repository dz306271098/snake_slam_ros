/**
 * Copyright (c) 2021 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "saiga/cuda/cusparseHelper.h"
#include "saiga/core/util/assert.h"

namespace Saiga
{
namespace CUDA
{
#ifdef SAIGA_USE_CUSPARSE

hipsparseHandle_t cusparseHandle = 0;
hipblasHandle_t cublashandle     = 0;

void initBLASSPARSE()
{
    if (!isBLASSPARSEInitialized())
    {
        hipblasCreate(&cublashandle);
        hipsparseCreate(&cusparseHandle);
    }
}

void destroyBLASSPARSE()
{
    if (isBLASSPARSEInitialized())
    {
        hipsparseDestroy(cusparseHandle);
        hipblasDestroy(cublashandle);
        cusparseHandle = 0;
        cublashandle   = 0;
    }
}

bool isBLASSPARSEInitialized()
{
    return cusparseHandle != 0;
}

extern void testCuBLAS();
extern void testCuSparse();
void runBLASSPARSETests()
{
    testCuBLAS();
    testCuSparse();
}

#endif

}  // namespace CUDA
}  // namespace Saiga
