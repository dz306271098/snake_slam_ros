#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2021 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "saiga/cuda/cudaHelper.h"
#include "saiga/cuda/cusparseHelper.h"
#include "saiga/cuda/tests/test.h"
#include "saiga/cuda/tests/test_helper.h"
#include "saiga/cuda/thread_info.h"
#include "saiga/core/time/timer.h"

namespace Saiga
{
namespace CUDA
{
#ifdef SAIGA_USE_CUSPARSE

void testCuSparse()
{
    //    0  3 0  0 0
    //    22 0 0  0 17
    //    7  5 0  1 0
    //    0  0 0  0 0
    //    0  0 14 0 8

    // in column major
    std::vector<double> denseMatrix = {0, 22, 7, 0, 0, 3, 0, 5, 0, 0, 0, 0, 0, 0, 14, 0, 0, 1, 0, 0, 0, 17, 0, 0, 8};

    std::vector<double> denseVector = {1, 2, 3, 4, 5};

    // result of the matrix vector product
    std::vector<double> ytarget = {6, 107, 21, 0, 82};


    std::vector<double> values = {22, 7, 3, 5, 14, 1, 17, 8};

    std::vector<int> rowIndx = {1, 2, 0, 2, 4, 2, 1, 4};

    std::vector<int> colPtr = {0, 2, 4, 5, 6, 8};

    thrust::device_vector<double> d_values = values;
    thrust::device_vector<int> d_rowIndx   = rowIndx;
    thrust::device_vector<int> d_colPtr    = colPtr;
    thrust::device_vector<double> d_x      = denseVector;
    thrust::device_vector<double> d_y(denseVector.size(), 0);

    hipsparseMatDescr_t mat;
    hipsparseCreateMatDescr(&mat);

    double alpha      = 1;
    const double beta = 2;

    hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE, 5, 5, values.size(), &alpha, mat,
                   thrust::raw_pointer_cast(d_values.data()), thrust::raw_pointer_cast(d_colPtr.data()),
                   thrust::raw_pointer_cast(d_rowIndx.data()), thrust::raw_pointer_cast(d_x.data()), &beta,
                   thrust::raw_pointer_cast(d_y.data()));

    thrust::host_vector<double> y = d_y;
    //    for(double d : y){
    //        std::cout << d << " ";
    //    }
    //    std::cout << std::endl;


    SAIGA_ASSERT(y == ytarget);



    std::cout << "cuSPARSE test: SUCCESS!" << std::endl;
}


/* Matrix size */
#    define N (275)

/* Host implementation of a simple version of sgemm */
static void simple_sgemm(int n, float alpha, const float* A, const float* B, float beta, float* C)
{
    int i;
    int j;
    int k;

    for (i = 0; i < n; ++i)
    {
        for (j = 0; j < n; ++j)
        {
            float prod = 0;

            for (k = 0; k < n; ++k)
            {
                prod += A[k * n + i] * B[j * n + k];
            }

            C[j * n + i] = alpha * prod + beta * C[j * n + i];
        }
    }
}


void testCuBLAS()
{
    hipblasStatus_t status;
    float* h_A;
    float* h_B;
    float* h_C;
    float* h_C_ref;
    float* d_A  = 0;
    float* d_B  = 0;
    float* d_C  = 0;
    float alpha = 1.0f;
    float beta  = 0.0f;
    int n2      = N * N;
    int i;
    float error_norm;
    float ref_norm;
    float diff;



    /* Allocate host memory for the matrices */
    h_A = (float*)malloc(n2 * sizeof(h_A[0]));

    if (h_A == 0)
    {
        fprintf(stderr, "!!!! host memory allocation error (A)\n");
        SAIGA_ASSERT(0);
    }

    h_B = (float*)malloc(n2 * sizeof(h_B[0]));

    if (h_B == 0)
    {
        fprintf(stderr, "!!!! host memory allocation error (B)\n");
        SAIGA_ASSERT(0);
    }

    h_C = (float*)malloc(n2 * sizeof(h_C[0]));

    if (h_C == 0)
    {
        fprintf(stderr, "!!!! host memory allocation error (C)\n");
        SAIGA_ASSERT(0);
    }

    /* Fill the matrices with test data */
    for (i = 0; i < n2; i++)
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
        h_C[i] = rand() / (float)RAND_MAX;
    }

    /* Allocate device memory for the matrices */
    if (hipMalloc((void**)&d_A, n2 * sizeof(d_A[0])) != hipSuccess)
    {
        fprintf(stderr, "!!!! device memory allocation error (allocate A)\n");
        SAIGA_ASSERT(0);
    }

    if (hipMalloc((void**)&d_B, n2 * sizeof(d_B[0])) != hipSuccess)
    {
        fprintf(stderr, "!!!! device memory allocation error (allocate B)\n");
        SAIGA_ASSERT(0);
    }

    if (hipMalloc((void**)&d_C, n2 * sizeof(d_C[0])) != hipSuccess)
    {
        fprintf(stderr, "!!!! device memory allocation error (allocate C)\n");
        SAIGA_ASSERT(0);
    }

    /* Initialize the device matrices with the host matrices */
    status = hipblasSetVector(n2, sizeof(h_A[0]), h_A, 1, d_A, 1);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "!!!! device access error (write A)\n");
        SAIGA_ASSERT(0);
    }

    status = hipblasSetVector(n2, sizeof(h_B[0]), h_B, 1, d_B, 1);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "!!!! device access error (write B)\n");
        SAIGA_ASSERT(0);
    }

    status = hipblasSetVector(n2, sizeof(h_C[0]), h_C, 1, d_C, 1);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "!!!! device access error (write C)\n");
        SAIGA_ASSERT(0);
    }

    /* Performs operation using plain C code */
    simple_sgemm(N, alpha, h_A, h_B, beta, h_C);
    h_C_ref = h_C;

    /* Performs operation using cublas */
    status = hipblasSgemm(cublashandle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "!!!! kernel execution error.\n");
        SAIGA_ASSERT(0);
    }

    /* Allocate host memory for reading back the result from device memory */
    h_C = (float*)malloc(n2 * sizeof(h_C[0]));

    if (h_C == 0)
    {
        fprintf(stderr, "!!!! host memory allocation error (C)\n");
        SAIGA_ASSERT(0);
    }

    /* Read the result back */
    status = hipblasGetVector(n2, sizeof(h_C[0]), d_C, 1, h_C, 1);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "!!!! device access error (read C)\n");
        SAIGA_ASSERT(0);
    }

    /* Check result against reference */
    error_norm = 0;
    ref_norm   = 0;

    for (i = 0; i < n2; ++i)
    {
        diff = h_C_ref[i] - h_C[i];
        error_norm += diff * diff;
        ref_norm += h_C_ref[i] * h_C_ref[i];
    }

    error_norm = (float)sqrt((double)error_norm);
    ref_norm   = (float)sqrt((double)ref_norm);

    if (fabs(ref_norm) < 1e-7)
    {
        fprintf(stderr, "!!!! reference norm is 0\n");
        SAIGA_ASSERT(0);
    }

    /* Memory clean up */
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_ref);

    if (hipFree(d_A) != hipSuccess)
    {
        fprintf(stderr, "!!!! memory free error (A)\n");
        SAIGA_ASSERT(0);
    }

    if (hipFree(d_B) != hipSuccess)
    {
        fprintf(stderr, "!!!! memory free error (B)\n");
        SAIGA_ASSERT(0);
    }

    if (hipFree(d_C) != hipSuccess)
    {
        fprintf(stderr, "!!!! memory free error (C)\n");
        SAIGA_ASSERT(0);
    }


    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "!!!! shutdown error (A)\n");
        SAIGA_ASSERT(0);
    }

    if (error_norm / ref_norm < 1e-6f)
    {
        std::cout << "cuBLAS test: SUCCESS!" << std::endl;
    }
    else
    {
        printf("simpleCUBLAS test failed.\n");
        SAIGA_ASSERT(0);
    }
}

#endif

}  // namespace CUDA
}  // namespace Saiga
