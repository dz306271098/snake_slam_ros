#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2021 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "saiga/core/time/timer.h"
#include "saiga/cuda/cudaHelper.h"
#include "saiga/cuda/scan.h"
#include "saiga/cuda/tests/test.h"
#include "saiga/cuda/tests/test_helper.h"
#include "saiga/cuda/thread_info.h"

namespace Saiga
{
namespace CUDA
{
using uint = unsigned int;

void scanTest()
{
    CUDA_SYNC_CHECK_ERROR();

    const bool exclusive           = false;
    const size_t THREADS_PER_BLOCK = 256;
    const int TILES_PER_BLOCK      = 8;
    const int ELEMENTS_PER_VECTOR  = 4;

    const int ELEMENTS_PER_BLOCK = THREADS_PER_BLOCK * TILES_PER_BLOCK * ELEMENTS_PER_VECTOR;
    int N                        = 100 * 1000 * 1000;

    std::cout << "Elements: " << N << " Elements per block: " << ELEMENTS_PER_BLOCK << std::endl;

    size_t readWrites = N * 2 * sizeof(uint);

    CUDA::PerformanceTestHelper pth("Scan (exclusive)", readWrites);


    thrust::host_vector<uint> h(N, 1);

    for (int i = 0; i < N; ++i)
    {
        h[i] = rand() % 4;
    }

    thrust::device_vector<uint> v = h;

    thrust::device_vector<uint> d_res(N + ELEMENTS_PER_BLOCK, -1);
    thrust::host_vector<uint> h_res(N + ELEMENTS_PER_BLOCK, -1);

    thrust::device_vector<uint> aggregate(CUDA::getBlockCount(N, ELEMENTS_PER_BLOCK) + 1, -1);

    {
        float time;
        {
            Saiga::ScopedTimer<float> t(&time);
            if (exclusive)
            {
                int sum = 0;
                for (int i = 0; i < N; ++i)
                {
                    h_res[i] = sum;
                    sum += h[i];
                }
            }
            else
            {
                int sum = 0;
                for (int i = 0; i < N; ++i)
                {
                    sum += h[i];
                    h_res[i] = sum;
                }
            }
        }
        pth.addMeassurement("CPU scan", time);
    }


    {
        float time;
        {
            CUDA::ScopedTimer t(time);
            if (exclusive)
            {
                thrust::exclusive_scan(v.begin(), v.end(), d_res.begin());
            }
            else
            {
                thrust::inclusive_scan(v.begin(), v.end(), d_res.begin());
            }
        }
        //        SAIGA_ASSERT(sum == res);
        pth.addMeassurement("thrust::scan", time);
    }

    SAIGA_ASSERT(d_res == h_res);



    {
        d_res = thrust::device_vector<uint>(N + ELEMENTS_PER_BLOCK, -1);

        float time;

        auto NUM_BLOCKS = CUDA::getBlockCount(N, ELEMENTS_PER_BLOCK);
        {
            CUDA::ScopedTimer t(time);
            CUDA::tiledSinglePassScan<exclusive, THREADS_PER_BLOCK, TILES_PER_BLOCK, int4, true>
                <<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(v, d_res, aggregate);
        }
        pth.addMeassurement("tiledSinglePassScan", time);
    }


    /*
    //check if the aggregate was computed correctly
    thrust::host_vector<unsigned int> h_a = aggregate;
    int i = ELEMENTS_PER_BLOCK;
    for(int ag : h_a){
        //        SAIGA_ASSERT(ag == i);
        i += ELEMENTS_PER_BLOCK;
    }

    thrust::host_vector<unsigned int> h_res2 = d_res;

    int maxPrint = ELEMENTS_PER_BLOCK * 2;
    for(int i = 0 ; i < int(h_res.size()) ; ++i){
        if(h_res2[i] != h_res[i]){
            std::cout << i << " " << h_res2[i] << "!=" << h_res[i] << std::endl;
            maxPrint--;
            if(maxPrint < 0)
                break;
        }
    }
    */

    SAIGA_ASSERT(d_res == h_res);


    {
        float time;
        {
            CUDA::ScopedTimer t(time);
            hipMemcpy(thrust::raw_pointer_cast(d_res.data()), thrust::raw_pointer_cast(v.data()), N * sizeof(int),
                       hipMemcpyDeviceToDevice);
        }
        pth.addMeassurement("hipMemcpy", time);
    }
    CUDA_SYNC_CHECK_ERROR();
}

}  // namespace CUDA
}  // namespace Saiga
