#include "hip/hip_runtime.h"
﻿/**
 * Copyright (c) 2021 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "saiga/cuda/cudaHelper.h"
#include "saiga/cuda/imageProcessing/imageProcessing.h"
#include "saiga/cuda/tests/test.h"
#include "saiga/cuda/tests/test_helper.h"

namespace Saiga
{
namespace CUDA
{
static void checkRes(ImageView<float> ref, ImageView<float> dst)
{
    for (int y = 0; y < ref.rows; ++y)
    {
        for (int x = 0; x < ref.cols; ++x)
        {
#if 0
            cout << dst(y,x) << " ";
#else
            if (std::abs(dst(y, x) - ref(y, x)) > 1e-5)
            {
                std::cout << "error (" << x << "," << y << ") " << dst(y, x) << "!=" << ref(y, x) << std::endl;
                SAIGA_ASSERT(0);
            }
#endif
        }
        //        cout << endl;
    }
}

// static void checkRes2(const thrust::host_vector<float>& ref, const thrust::host_vector<float>& dst){
//    int c = 0;
//    for(int i = 0; i < (int)dst.size();++i){
//        auto refv = 9.0f;
//        if(std::abs(dst[i] - refv) > 1e-5){
//            cout << "error " << (i/2048) << "," << (i%2048) << " " << dst[i] << "!=" << refv << endl;
//            c++;
//            SAIGA_ASSERT(c < 5);
//        }
//    }
//}

template <int KERNEL_RADIUS>
void convolutionTest2(int w, int h)
{
    CUDA_SYNC_CHECK_ERROR();

    size_t N          = w * h;
    size_t readWrites = N * 2 * sizeof(float);


    size_t pitch = Saiga::iAlignUp(sizeof(float) * w, 1024 * 1024);
    size_t size  = pitch * h;


    thrust::device_vector<char> src(size, 0);
    thrust::device_vector<char> dest(size, 0);
    thrust::device_vector<char> tmp(size, 0);

    thrust::host_vector<char> h_src  = src;
    thrust::host_vector<char> h_dest = dest;
    thrust::host_vector<char> h_tmp  = dest;
    thrust::host_vector<char> h_ref  = dest;

    ImageView<float> imgSrc(h, w, pitch, thrust::raw_pointer_cast(src.data()));
    ImageView<float> imgDst(h, w, pitch, thrust::raw_pointer_cast(dest.data()));
    ImageView<float> imgTmp(h, w, pitch, thrust::raw_pointer_cast(tmp.data()));


    ImageView<float> h_imgSrc(h, w, pitch, thrust::raw_pointer_cast(h_src.data()));
    ImageView<float> h_imgDst(h, w, pitch, thrust::raw_pointer_cast(h_dest.data()));
    ImageView<float> h_imgTmp(h, w, pitch, thrust::raw_pointer_cast(h_tmp.data()));

    int its = 50;
    //    float sigma = 2.0f;
    //    thrust::device_vector<float> d_kernel = createGaussianBlurKernel(KERNEL_RADIUS,sigma);
    thrust::device_vector<float> d_kernel(2 * KERNEL_RADIUS + 1, 1.0f);

    thrust::host_vector<float> h_kernel(d_kernel);

    {
        for (int y = 0; y < h; ++y)
        {
            for (int x = 0; x < w; ++x)
            {
                //                h_imgSrc(y,x) = (rand()%3) - 1;
                h_imgSrc(y, x) = 1;
            }
        }
        src = h_src;
    }

#if 0
//    cout << "first pixels: " << h_imgSrc(0,0) << " " << h_imgSrc(0,1) << " " << h_imgSrc(1,0) << " " << h_imgSrc(1,1) << endl;


    int fx = 508 % w;
    int fy = 508 / w;
    cout << "debug pixel: " << fx << " " << fy << endl;
    for (int j=-KERNEL_RADIUS;j<=KERNEL_RADIUS;j++){
                float sum =0;
                            for (int i=-KERNEL_RADIUS;i<=KERNEL_RADIUS;i++){
                                auto v = h_imgSrc.clampedRead(fy+j,fx+i);
                                sum += v;
                               cout << v << " ";
                            }
                            cout << " row sum: " << sum << endl;
    }
#endif

    Saiga::CUDA::PerformanceTestHelper pth("convolutionTest radius=" + std::to_string(KERNEL_RADIUS) +
                                               " ImageSize: " + std::to_string(w) + "x" + std::to_string(h),
                                           readWrites);

    // this takes too long :D
#if 0
    {
        float time;
        {
            Saiga::ScopedTimer<float> t(&time);
            for(int y = 0; y < h; ++y){
                for(int x = 0; x < w; ++x){
                    float sum = 0;
                    for (int j=-KERNEL_RADIUS;j<=KERNEL_RADIUS;j++){
                        float innerSum = 0;
                        for (int i=-KERNEL_RADIUS;i<=KERNEL_RADIUS;i++){
                            innerSum += h_imgSrc.clampedRead(y +j ,x + i) * h_kernel[i+KERNEL_RADIUS];
                        }
                        sum += innerSum * h_kernel[j+KERNEL_RADIUS];
                    }
                    h_imgDst(y,x) = sum;
                }
            }
        }
        pth.addMeassurement("CPU Convolve",time);
        h_ref = h_dest;
    }
#endif

    {
        float time;
        {
            Saiga::ScopedTimer<float> t(&time);
            for (int y = 0; y < h; ++y)
            {
                for (int x = 0; x < w; ++x)
                {
                    float sum = 0;
                    for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
                    {
                        sum += h_imgSrc.clampedRead(y, x + j) * h_kernel[j + KERNEL_RADIUS];
                    }
                    h_imgTmp(y, x) = sum;
                }
            }

            for (int x = 0; x < w; ++x)
            {
                for (int y = 0; y < h; ++y)
                {
                    float sum = 0;
                    for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
                    {
                        sum += h_imgTmp.clampedRead(y + j, x) * h_kernel[j + KERNEL_RADIUS];
                    }
                    h_imgDst(y, x) = sum;
                }
            }
        }
        pth.addMeassurement("CPU Convolve Separate", time);
        h_ref = h_dest;
    }


#if 0
    {
        thrust::device_vector<float> d_kernel = h_kernel;
        dest = src;
        auto st = Saiga::measureObject<Saiga::CUDA::CudaScopedTimer>(its, [&]()
        {
            convolveSinglePassSeparateOuterLinear(imgSrc,imgDst,d_kernel,KERNEL_RADIUS);
        });


        pth.addMeassurement("convolveSinglePassSeparateOuterLinear",st.median);
        checkRes(h_ref,thrust::host_vector<float>(dest));
    }

    {
        thrust::device_vector<float> d_kernel = h_kernel;
        dest = src;


        auto st = Saiga::measureObject<Saiga::CUDA::CudaScopedTimer>(its, [&]()
        {
            convolveSinglePassSeparateOuterHalo(imgSrc,imgDst,d_kernel,KERNEL_RADIUS);
        });
        pth.addMeassurement("convolveSinglePassSeparateOuterHalo",st.median);
        checkRes(h_ref,thrust::host_vector<float>(dest));
    }

    {
        thrust::device_vector<float> d_kernel = h_kernel;
        dest = src;
        auto st = Saiga::measureObject<Saiga::CUDA::CudaScopedTimer>(its, [&]()
        {
            convolveSinglePassSeparateInner(imgSrc,imgDst,d_kernel,KERNEL_RADIUS);
        });
        pth.addMeassurement("convolveSinglePassSeparateInner",st.median);
        checkRes(h_ref,thrust::host_vector<float>(dest));
    }

#endif
    {
        thrust::device_vector<float> d_kernel = h_kernel;
        dest                                  = src;
        auto st                               = Saiga::measureObject<Saiga::CUDA::ScopedTimer>(
            its, [&]() { convolveSinglePassSeparateInner75(imgSrc, imgDst, d_kernel, KERNEL_RADIUS); });
        pth.addMeassurement("convolveSinglePassSeparateInner75", st.median);
        // checkRes(h_ref,thrust::host_vector<float>(dest));
    }

    CUDA_SYNC_CHECK_ERROR();

    {
        thrust::device_vector<float> d_kernel = h_kernel;
        //        dest = src;
        thrust::fill(dest.begin(), dest.end(), 0.0f);
        auto st = Saiga::measureObject<Saiga::CUDA::ScopedTimer>(
            its, [&]() { convolveSinglePassSeparateInnerShuffle(imgSrc, imgDst, d_kernel, KERNEL_RADIUS); });
        pth.addMeassurement("convolveSinglePassSeparateInnerShuffle", st.median);

        thrust::host_vector<char> d(dest);
        ImageView<float> i(h, w, pitch, thrust::raw_pointer_cast(d.data()));
        checkRes(h_imgDst, i);
        //        checkRes2(h_ref,thrust::host_vector<float>(dest));
    }

    CUDA_SYNC_CHECK_ERROR();


#if 1
    {
        dest                                  = src;
        tmp                                   = src;
        thrust::device_vector<float> d_kernel = h_kernel;

        auto st1 = Saiga::measureObject<Saiga::CUDA::ScopedTimer>(
            its, [&]() { convolveRow(imgSrc, imgTmp, d_kernel, KERNEL_RADIUS); });
        pth.addMeassurement("GPU Convolve Separate Row", st1.median);

        auto st2 = Saiga::measureObject<Saiga::CUDA::ScopedTimer>(
            its, [&]() { convolveCol(imgTmp, imgDst, d_kernel, KERNEL_RADIUS); });
        pth.addMeassurement("GPU Convolve Separate Col", st2.median);
        pth.addMeassurement("GPU Convolve Separate Total", st1.median + st2.median);

        //        checkRes(h_ref,thrust::host_vector<float>(dest));
    }
#endif

    {
        auto st = Saiga::measureObject<Saiga::CUDA::ScopedTimer>(its, [&]() {
            hipMemcpy(thrust::raw_pointer_cast(dest.data()), thrust::raw_pointer_cast(src.data()), N * sizeof(int),
                       hipMemcpyDeviceToDevice);
        });
        pth.addMeassurement("hipMemcpy", st.median);
    }
    CUDA_SYNC_CHECK_ERROR();
}

void convolutionTest()
{
    //    convolutionTest2<3>(17,53);
    int w = 2048;
    int h = 1024;
    //    int w = 512;
    //    int h = 256;


    //    convolutionTest2<1>(w,h);
    //    convolutionTest2<2>(w,h);
    //    convolutionTest2<3>(w,h);
    convolutionTest2<4>(w, h);
    //    convolutionTest2<5>(w,h);
    //    convolutionTest2<6>(w,h);
    //    convolutionTest2<7>(w,h);
    //    convolutionTest2<8>(w,h);

    //    convolutionTest2<9>(w,h);
    //    convolutionTest2<10>(w,h);
    //    convolutionTest2<11>(w,h);
    //    convolutionTest2<12>(w,h);
    //    convolutionTest2<13>(w,h);
    //    convolutionTest2<14>(w,h);
    //    convolutionTest2<15>(w,h);
    //    convolutionTest2<16>(w,h);
}

}  // namespace CUDA
}  // namespace Saiga
